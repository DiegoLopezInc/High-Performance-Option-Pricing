#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_kernels.h"

__global__ void blackScholesKernel(const OptionData* options, double* results, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        OptionData opt = options[idx];
        double d1 = (log(opt.spot / opt.strike) + (opt.riskFreeRate + 0.5 * opt.volatility * opt.volatility) * opt.maturity) / (opt.volatility * sqrt(opt.maturity));
        double d2 = d1 - opt.volatility * sqrt(opt.maturity);
        results[idx] = opt.spot * normcdf(d1) - opt.strike * exp(-opt.riskFreeRate * opt.maturity) * normcdf(d2);
    }
}

void launchBlackScholesKernel(const OptionData* h_options, double* h_results, int n) {
    OptionData* d_options;
    double* d_results;

    hipMalloc(&d_options, n * sizeof(OptionData));
    hipMalloc(&d_results, n * sizeof(double));

    hipMemcpy(d_options, h_options, n * sizeof(OptionData), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    blackScholesKernel<<<numBlocks, blockSize>>>(d_options, d_results, n);

    hipMemcpy(h_results, d_results, n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_options);
    hipFree(d_results);
}
